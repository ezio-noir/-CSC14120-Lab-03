#include "hip/hip_runtime.h"
#include <stdio.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start,0);
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

__global__ void addVecKernel(int *in1, int *in2, int n, 
        int *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    if (i < n)
    {
        out[i] = in1[i] + in2[i];
    }
}

void addVec(int *in1, int *in2, int n, 
        int *out, 
        bool useDevice=false, dim3 blockSize=dim3(1), int nStreams=1)
{
	if (useDevice == false)
	{
        for (int i = 0; i < n; i++)
        {
            out[i] = in1[i] + in2[i];
        }
	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);
        
        size_t nBytes = n * sizeof(int);

		// TODO: Allocate device memory regions
		int *d_in1, *d_in2, *d_out;
		CHECK(hipMalloc((void**) &d_in1, nBytes));
		CHECK(hipMalloc((void**) &d_in2, nBytes));
		CHECK(hipMalloc((void**) &d_out, nBytes));

        // TODO: Create "nStreams" device streams
		hipStream_t *streams;
		streams = (hipStream_t*)malloc(nStreams * sizeof(hipStream_t));
		if (!streams)
		{
			fprintf(stderr, "Cannot allocate memory.\n");
			exit(EXIT_FAILURE);
		}
		for (int i = 0; i < nStreams; ++i)
		{
			CHECK(hipStreamCreate(&streams[i]));
		}

        GpuTimer timer;
        timer.Start();

        // TODO: Send jobs (H2D, kernel, D2H) to device streams 
		int nBlocks = (n - 1) / blockSize.x + 1;
		int maxNBlocksPerStream = (nBlocks - 1) / nStreams + 1;
		for (int i = 0; i < nStreams; ++i)
		{
			int blockOffset = i * maxNBlocksPerStream;
			int nBlocksStream = min(maxNBlocksPerStream, nBlocks - blockOffset);
			if (nBlocksStream <= 0) break;
			int offsetElem = blockOffset * blockSize.x;
			int nElemsStream = n - offsetElem;

			CHECK(hipMemcpyAsync(d_in1 + offsetElem, in1 + offsetElem, nElemsStream * sizeof(int), hipMemcpyHostToDevice, streams[i]));
			CHECK(hipMemcpyAsync(d_in2 + offsetElem, in2 + offsetElem, nElemsStream * sizeof(int), hipMemcpyHostToDevice, streams[i]));
			addVecKernel<<<dim3(nBlocksStream), blockSize, 0, streams[i]>>>(d_in1 + offsetElem, d_in2 + offsetElem, nElemsStream, d_out + offsetElem);
			CHECK(hipMemcpyAsync(out + offsetElem, d_out + offsetElem, nElemsStream * sizeof(int), hipMemcpyDeviceToHost, streams[i]));
		}
		// Wait for nStreams streams to finish
		for (int i = 0; i < nStreams; ++i) CHECK(hipStreamSynchronize(streams[i]));

        timer.Stop();
        float time = timer.Elapsed();
        printf("Processing time of all device streams: %f ms\n\n", time);

        // TODO: Destroy device streams
		for (int i = 0; i < nStreams; ++i)
		{
			CHECK(hipStreamDestroy(streams[i]));
		}
		free(streams);

        // TODO: Free device memory regions
		CHECK(hipFree(d_in1));
		CHECK(hipFree(d_in2));
		CHECK(hipFree(d_out));
	}
}

int main(int argc, char ** argv)
{
    int n; 
    int *in1, *in2; 
    int *out, *correctOut;

    // Input data into n
    n = (1 << 24) + 1;
    printf("n =  %d\n\n", n);

    // Allocate memories for in1, in2, out
    size_t nBytes = n * sizeof(int);
    CHECK(hipHostMalloc(&in1, nBytes));
    CHECK(hipHostMalloc(&in2, nBytes));
    CHECK(hipHostMalloc(&out, nBytes));
    correctOut = (int *)malloc(nBytes);

    // Input data into in1, in2
    for (int i = 0; i < n; i++)
    {
    	in1[i] = rand() & 0xff; // Random int in [0, 255]
    	in2[i] = rand() & 0xff; // Random int in [0, 255]
    }

    // Add in1 & in2 on host
    addVec(in1, in2, n, correctOut);

    // Add in1 & in2 on device
	dim3 blockSize(512); // Default
    int nStreams = 1; // Default
	if (argc >= 2)
	{
		blockSize.x = atoi(argv[1]);
        if (argc >= 3)
        {
            nStreams = atoi(argv[2]);
        }
	} 
    addVec(in1, in2, n, out, true, blockSize, nStreams);

    // Check correctness
    for (int i = 0; i < n; i++)
    {
    	if (out[i] != correctOut[i])
    	{
    		printf("INCORRECT :(\n");
    		return 1;
    	}
    }
    printf("CORRECT :)\n");
    
    CHECK(hipHostFree(in1));
    CHECK(hipHostFree(in2));
    CHECK(hipHostFree(out));    
    free(correctOut);
}
