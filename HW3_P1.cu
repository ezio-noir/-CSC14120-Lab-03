
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define FILTER_WIDTH 9
__constant__ float dc_filter[FILTER_WIDTH * FILTER_WIDTH];

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height, char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

__global__ void blurImgKernel1(uchar3 * inPixels, int width, int height, 
        float * filter, int filterWidth, 
        uchar3 * outPixels)
{
	// TODO
	int o_r = blockIdx.y * blockDim.y + threadIdx.y;
	int o_c = blockIdx.x * blockDim.x + threadIdx.x;
	if (o_r >= 0 && o_r < width && o_c >= 0 && o_c < height)
	{
		int i_r, i_c;
		int filterRadius = filterWidth / 2;
		uchar3 inPixel;
		float valX, valY, valZ;
		valX = valY = valZ = 0;
		for (int f_r = 0; f_r < filterWidth; ++f_r)
		{
			for (int f_c = 0; f_c < filterWidth; ++f_c)
			{
				i_r = o_r + f_r - filterRadius;
				i_c = o_c + f_c - filterRadius;
				if (i_r >= 0 && i_r < width && i_c >= 0 && i_c < height)
				{
					inPixel = inPixels[i_r * width + i_c];
					valX += filter[f_r * filterWidth + f_c] * static_cast<float>(inPixel.x);
					valY += filter[f_r * filterWidth + f_c] * static_cast<float>(inPixel.y);
					valZ += filter[f_r * filterWidth + f_c] * static_cast<float>(inPixel.z);
				}
			}
		}
		outPixels[o_r * width + o_c] = make_uchar3(static_cast<unsigned char>(valX), static_cast<unsigned char>(valY), static_cast<unsigned char>(valZ));
	}
}

__global__ void blurImgKernel2(uchar3 * inPixels, int width, int height, 
        float * filter, int filterWidth, 
        uchar3 * outPixels)
{
	// TODO
	extern __shared__ uchar3 s_inPixels[];

	int filterRadius =  filterWidth / 2;
	dim3 inTileSize(blockDim.x + 2 * filterRadius, blockDim.y + 2 * filterRadius); // size of input tile
	int numInTileElems = inTileSize.x * inTileSize.y;
	int numOutTileElems = blockDim.x * blockDim.y;

	int idx1DToLoad; // index in s_inPixels
	int i_r, i_c; // row and column in input matrix

	// Top-left pixel of the tile in the input matrix
	uint2 inTopLeft = make_uint2(blockIdx.x * blockDim.x - filterRadius, blockIdx.y * blockDim.y - filterRadius);

	// Load input tile into shared memory
	// Explaination for this approach in report (notebook file)
	for (int i = 0; i < (numInTileElems - 1) / numOutTileElems + 1; ++i)
	{
		idx1DToLoad = (i * numOutTileElems) + (threadIdx.y * blockDim.x + threadIdx.x);
		if (idx1DToLoad >= numInTileElems)
		{
			break;
		}
		i_r = inTopLeft.y + idx1DToLoad / inTileSize.x;
		i_c = inTopLeft.x + idx1DToLoad % inTileSize.x;
		s_inPixels[idx1DToLoad] = (i_r >= 0 && i_r < height && i_c >= 0 && i_c < width)
			? inPixels[i_r * width + i_c]
			: make_uchar3(0, 0, 0);
	}
	__syncthreads();

	int o_r = blockIdx.y * blockDim.y + threadIdx.y;
	int o_c = blockIdx.x * blockDim.x + threadIdx.x;
	int s_r, s_c; // Row and column of input tile (in shared memory)
	if (o_r >= 0 && o_r < height && o_c >= 0 && o_c < width)
	{
		float valX, valY, valZ;
		valX = valY = valZ = 0;
		for (int f_r = 0; f_r < filterWidth; ++f_r)
		{
			for (int f_c = 0; f_c < filterWidth; ++f_c)
			{
				s_r = threadIdx.y + f_r;
				s_c = threadIdx.x + f_c;
				valX += filter[f_r * filterWidth + f_c] * static_cast<float>(s_inPixels[s_r * inTileSize.x + s_c].x);
				valY += filter[f_r * filterWidth + f_c] * static_cast<float>(s_inPixels[s_r * inTileSize.x + s_c].y);
				valZ += filter[f_r * filterWidth + f_c] * static_cast<float>(s_inPixels[s_r * inTileSize.x + s_c].z);
			}
		}
		outPixels[o_r * width + o_c] = make_uchar3(static_cast<unsigned char>(valX), static_cast<unsigned char>(valY), static_cast<unsigned char>(valZ));
	}
}

__global__ void blurImgKernel3(uchar3 * inPixels, int width, int height, 
        int filterWidth, 
        uchar3 * outPixels)
{
	// TODO
	extern __shared__ uchar3 s_inPixels[];

	int filterRadius =  filterWidth / 2;
	dim3 inTileSize(blockDim.x + 2 * filterRadius, blockDim.y + 2 * filterRadius); // size of input tile
	int numInTileElems = inTileSize.x * inTileSize.y;
	int numOutTileElems = blockDim.x * blockDim.y;

	int idx1DToLoad; // index in s_inPixels
	int i_r, i_c; // row and column in input matrix

	// Top-left pixel of the tile in the input
	uint2 inTopLeft = make_uint2(blockIdx.x * blockDim.x - filterRadius, blockIdx.y * blockDim.y - filterRadius);

	// Load input tile into shared memory
	// Explaination for this approach in report (notebook file)
	for (int i = 0; i < (numInTileElems - 1) / numOutTileElems + 1; ++i)
	{
		idx1DToLoad = (i * numOutTileElems) + (threadIdx.y * blockDim.x + threadIdx.x);
		if (idx1DToLoad >= numInTileElems)
		{
			break;
		}
		i_r = inTopLeft.y + idx1DToLoad / inTileSize.x;
		i_c = inTopLeft.x + idx1DToLoad % inTileSize.x;
		s_inPixels[idx1DToLoad] = (i_r >= 0 && i_r < height && i_c >= 0 && i_c < width)
			? inPixels[i_r * width + i_c]
			: make_uchar3(0, 0, 0);
	}
	__syncthreads();

	int o_r = blockIdx.y * blockDim.y + threadIdx.y;
	int o_c = blockIdx.x * blockDim.x + threadIdx.x;
	int s_r, s_c; // Row and column of input tile (in shared memory)
	if (o_r >= 0 && o_r < height && o_c >= 0 && o_c < width)
	{
		float valX, valY, valZ;
		valX = valY = valZ = 0;
		for (int f_r = 0; f_r < filterWidth; ++f_r)
		{
			for (int f_c = 0; f_c < filterWidth; ++f_c)
			{
				s_r = threadIdx.y + f_r;
				s_c = threadIdx.x + f_c;
				valX += dc_filter[f_r * filterWidth + f_c] * static_cast<float>(s_inPixels[s_r * inTileSize.x + s_c].x);
				valY += dc_filter[f_r * filterWidth + f_c] * static_cast<float>(s_inPixels[s_r * inTileSize.x + s_c].y);
				valZ += dc_filter[f_r * filterWidth + f_c] * static_cast<float>(s_inPixels[s_r * inTileSize.x + s_c].z);
			}
		}
		outPixels[o_r * width + o_c] = make_uchar3(static_cast<unsigned char>(valX), static_cast<unsigned char>(valY), static_cast<unsigned char>(valZ));
	}
}							

void blurImg(uchar3 * inPixels, int width, int height, float * filter, int filterWidth, 
        uchar3 * outPixels,
        bool useDevice=false, dim3 blockSize=dim3(1, 1), int kernelType=1)
{
	if (useDevice == false)
	{
		//TODO
		int filterRadius = filterWidth / 2;
		float valX, valY, valZ;
		int i_r, i_c, o_r, o_c, f_r, f_c;
		for (o_r = 0; o_r < height; ++o_r)
		{
			for (o_c = 0; o_c < width; ++o_c)
			{
				valX = valY = valZ = 0;
				for (f_r = 0; f_r < filterWidth; ++f_r)
				{
					for (f_c = 0; f_c < filterWidth; ++f_c)
					{
						i_r = o_r + f_r - filterRadius; i_c = o_c + f_c - filterRadius;
						if (i_r >= 0 && i_r < height && i_c >= 0 && i_c < width)
						{
							valX += filter[f_r * filterWidth + f_c] * static_cast<float>(inPixels[i_r * width + i_c].x);
							valY += filter[f_r * filterWidth + f_c] * static_cast<float>(inPixels[i_r * width + i_c].y);
							valZ += filter[f_r * filterWidth + f_c] * static_cast<float>(inPixels[i_r * width + i_c].z);
						}
					}
				}
				outPixels[o_r * width + o_c] = make_uchar3(static_cast<unsigned char>(valX), static_cast<unsigned char>(valY), static_cast<unsigned char>(valZ));
			}
		}
	}
	else // Use device
	{
		GpuTimer timer;
		
		printf("\nKernel %i, ", kernelType);
		// Allocate device memories
		uchar3 * d_inPixels, * d_outPixels;
		float * d_filter;
		size_t pixelsSize = width * height * sizeof(uchar3);
		size_t filterSize = filterWidth * filterWidth * sizeof(float);
		CHECK(hipMalloc(&d_inPixels, pixelsSize));
		CHECK(hipMalloc(&d_outPixels, pixelsSize));
		if (kernelType == 1 || kernelType == 2)
		{
			CHECK(hipMalloc(&d_filter, filterSize));
		}

		// Copy data to device memories
		CHECK(hipMemcpy(d_inPixels, inPixels, pixelsSize, hipMemcpyHostToDevice));
		if (kernelType == 1 || kernelType == 2)
		{
			CHECK(hipMemcpy(d_filter, filter, filterSize, hipMemcpyHostToDevice));
		}
		else
		{
			// TODO: copy data from "filter" (on host) to "dc_filter" (on CMEM of device)
			CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_filter), filter, filterWidth * filterWidth * sizeof(float)));
		}

		// Call kernel
		dim3 gridSize((width-1)/blockSize.x + 1, (height-1)/blockSize.y + 1);
		printf("block size %ix%i, grid size %ix%i\n", blockSize.x, blockSize.y, gridSize.x, gridSize.y);
		timer.Start();
		if (kernelType == 1)
		{
			// TODO: call blurImgKernel1
			blurImgKernel1<<<gridSize, blockSize>>>(d_inPixels, width, height, d_filter, filterWidth, d_outPixels);
		}
		else if (kernelType == 2)
		{
			// TODO: call blurImgKernel2
			dim3 inTileSize(blockSize.x + filterWidth - 1, blockSize.y + filterWidth - 1);
			blurImgKernel2<<<gridSize, blockSize, inTileSize.x * inTileSize.y * sizeof(uchar3)>>>(d_inPixels, width, height, d_filter, filterWidth, d_outPixels);
		}
		else
		{
			// TODO: call blurImgKernel3
			dim3 inTileSize(blockSize.x + filterWidth - 1, blockSize.y + filterWidth - 1);
			blurImgKernel3<<<gridSize, blockSize, inTileSize.x * inTileSize.y * sizeof(uchar3)>>>(d_inPixels, width, height, filterWidth, d_outPixels);
		}
		timer.Stop();
		float time = timer.Elapsed();
		printf("Kernel time: %f ms\n", time);
		hipDeviceSynchronize();
		CHECK(hipGetLastError());

		// Copy result from device memory
		CHECK(hipMemcpy(outPixels, d_outPixels, pixelsSize, hipMemcpyDeviceToHost));

		// Free device memories
		CHECK(hipFree(d_inPixels));
		CHECK(hipFree(d_outPixels));
		if (kernelType == 1 || kernelType == 2)
		{
			CHECK(hipFree(d_filter));
		}
	}
	
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

void printError(uchar3 * deviceResult, uchar3 * hostResult, int width, int height)
{
	float err = computeError(deviceResult, hostResult, width * height);
	printf("Error: %f\n", err);
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);
    printf("****************************\n");

}

int main(int argc, char ** argv)
{
	if (argc !=3 && argc != 5)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	printDeviceInfo();

	// Read input image file
	int width, height;
	uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("\nImage size (width x height): %i x %i\n", width, height);

	// Set up a simple filter with blurring effect 
	int filterWidth = FILTER_WIDTH;
	float * filter = (float *)malloc(filterWidth * filterWidth * sizeof(float));
	for (int filterR = 0; filterR < filterWidth; filterR++)
	{
		for (int filterC = 0; filterC < filterWidth; filterC++)
		{
			filter[filterR * filterWidth + filterC] = 1. / (filterWidth * filterWidth);
		}
	}

	// Blur input image not using device
	uchar3 * correctOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3)); 
	blurImg(inPixels, width, height, filter, filterWidth, correctOutPixels);
	
    // Blur input image using device, kernel 1
    dim3 blockSize(16, 16); // Default
	if (argc == 5)
	{
		blockSize.x = atoi(argv[3]);
		blockSize.y = atoi(argv[4]);
	}	
	uchar3 * outPixels1 = (uchar3 *)malloc(width * height * sizeof(uchar3));
	blurImg(inPixels, width, height, filter, filterWidth, outPixels1, true, blockSize, 1);
	printError(outPixels1, correctOutPixels, width, height);
	
	// Blur input image using device, kernel 2
	uchar3 * outPixels2 = (uchar3 *)malloc(width * height * sizeof(uchar3));
	blurImg(inPixels, width, height, filter, filterWidth, outPixels2, true, blockSize, 2);
	printError(outPixels2, correctOutPixels, width, height);

	// Blur input image using device, kernel 3
	uchar3 * outPixels3 = (uchar3 *)malloc(width * height * sizeof(uchar3));
	blurImg(inPixels, width, height, filter, filterWidth, outPixels3, true, blockSize, 3);
	printError(outPixels3, correctOutPixels, width, height);

    // Write results to files
    char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(correctOutPixels, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(outPixels1, width, height, concatStr(outFileNameBase, "_device1.pnm"));
	writePnm(outPixels2, width, height, concatStr(outFileNameBase, "_device2.pnm"));
	writePnm(outPixels3, width, height, concatStr(outFileNameBase, "_device3.pnm"));

	// Free memories
	free(inPixels);
	free(filter);
	free(correctOutPixels);
	free(outPixels1);
	free(outPixels2);
	free(outPixels3);
}
